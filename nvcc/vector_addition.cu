#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1000000
#define BLOCK_SIZE 256

// Kernel definition
__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    size_t size = N * sizeof(float);

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++)
    {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Transfer data from host to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    int gridSize = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    VecAdd<<<gridSize, BLOCK_SIZE>>>(d_A, d_B, d_C);

    // Transfer results from device to host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verification
    for (int i = 0; i < N; i++)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(1);
        }
    }
    printf("Test PASSED\n");

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
