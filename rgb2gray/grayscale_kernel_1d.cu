#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAStream.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

__global__
void rgb_to_grayscale_kernel(unsigned char* x, unsigned char* out, int n) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i<n) out[i] = 0.2989*x[i] + 0.5870*x[i+n] + 0.1140*x[i+2*n];
}


// helper function for ceiling unsigned integer division
inline unsigned int cdiv(unsigned int a, unsigned int b) {
  return (a + b - 1) / b;
}


torch::Tensor rgb_to_grayscale(torch::Tensor input) {
    CHECK_INPUT(input);
    int h = input.size(1);
    int w = input.size(2);
    printf("h*w: %d*%d\n", h, w);
    auto output = torch::empty({h,w}, input.options());
    int threads = 256;
    rgb_to_grayscale_kernel<<<cdiv(w*h,threads), threads>>>(
        input.data_ptr<unsigned char>(), output.data_ptr<unsigned char>(), w*h);
    // check CUDA error status (calls hipGetLastError())
    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return output;
}
